#include "hip/hip_runtime.h"
//iojpegparts.cu

#include <stdio.h>
#include <malloc.h>
#include <stdlib.h>
#include <math.h>
#include <jpeglib.h>
#include <sys/time.h>

double cpuSecond(){
	struct timeval tp;
	gettimeofday(&tp,NULL);
	return((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

int usage(char *name){
	printf("Code to blur parts of image using GPUs.\n");
	printf("Usage as follows: %s InputFileName OutputFileName MaskWidth PeakWidth\n",name);
	exit(1);
}

__global__ void GaussianBlurCuda (unsigned char *pic, unsigned char * outpic, double *mask, int *size){ // size: width, height, mask_width
	int pxPosCen = blockIdx.x * blockDim.x + threadIdx.x;
	if (pxPosCen >= size[0]*size[1] || pxPosCen < 0) return;
	int row, col, x, y, pos;
	row = pxPosCen/size[0]; // pixel position taken as width major
	col = pxPosCen%size[0];
	double sumout[3];
	sumout[0] = 0;
	sumout[1] = 0;
	sumout[2] = 0;
	if (row < size[2]/2 || row >= (size[1] - (size[2]/2))) return;
	if (col < size[2]/2 || col >= (size[0] - (size[2]/2))) return;
	for (int i=0;i<size[2]*size[2];i++){
		x = i%size[2] + col - size[2]/2;
		y = i/size[2] + row - size[2]/2;
		pos = (y*size[0]  + x)*3;
		sumout[0]+=(double)(*(pic+pos  )) * mask[i];
		sumout[1]+=(double)(*(pic+pos+1)) * mask[i];
		sumout[2]+=(double)(*(pic+pos+2)) * mask[i];
	}
	pos = pxPosCen*3;
	*(outpic+pos) = (unsigned char) sumout[0];
	*(outpic+pos+1) = (unsigned char) sumout[1];
	*(outpic+pos+2) = (unsigned char) sumout[2];
}


int main (int argc, char *argv[]){
	if (argc != 5) usage(argv[0]);
	int width, height;
	char *name = argv[1];
	char *out = argv[2];
	int mask_width = atoi(argv[3]);
	double peak_width = atof(argv[4]);
	if (mask_width%2 !=1){
		printf("Mask width must be odd.\n");
		exit(1);
	}

	double tStart = cpuSecond();

	FILE *infile = fopen(name,"rb");
	FILE *outfile = fopen(out,"wb");
	if (infile == NULL){
		printf("Could not read file\n");
		return 1;
	}
	struct jpeg_decompress_struct cinfo;
	struct jpeg_compress_struct cinfo1;
	struct jpeg_error_mgr jerr;
	JSAMPARRAY pJpegBuffer;

	cinfo.err = jpeg_std_error(&jerr);
	jpeg_create_decompress(&cinfo);
	jpeg_stdio_src(&cinfo, infile);
	jpeg_read_header(&cinfo, TRUE);
	jpeg_start_decompress(&cinfo);
	int row_stride = cinfo.output_width * cinfo.output_components;
	width = cinfo.output_width;
	height = cinfo.output_height;

	unsigned char *pic, *outpic;
	pic = (unsigned char *) malloc(width*height*3*sizeof(pic));
	outpic = (unsigned char *) malloc(width*height*3*sizeof(outpic));
	pJpegBuffer = (*cinfo.mem->alloc_sarray) ((j_common_ptr) &cinfo, JPOOL_IMAGE, row_stride, 1);
	while (cinfo.output_scanline < cinfo.output_height) {
		(void) jpeg_read_scanlines(&cinfo, pJpegBuffer, 1);
		for (int x=0;x<width;x++) {
			*(pic++) = pJpegBuffer[0][cinfo.output_components*x];
			if (cinfo.output_components>2){
				*(pic++) = pJpegBuffer[0][cinfo.output_components*x+1];
				*(pic++) = pJpegBuffer[0][cinfo.output_components*x+2];
			} else {
				*(pic++) = *(pic-1);
				*(pic++) = *(pic-1);
			}
		}
	}
	pic -= width*height*3;

	fclose(infile);
	(void) jpeg_finish_decompress(&cinfo);
	jpeg_destroy_decompress(&cinfo);

	double * mask;
	mask = (double *) malloc(mask_width*mask_width*sizeof(mask));
	int x,y,xcen=mask_width/2,ycen=xcen;
	double a = 1/(peak_width*peak_width * 44/7), sum=0;
	for (int i=0;i<mask_width*mask_width;i++){
		x = i%mask_width;
		y = i/mask_width;
		mask[i] = a * exp(-(x-xcen)*(x-xcen)/(2*peak_width*peak_width)
					      -(y-ycen)*(y-ycen)/(2*peak_width*peak_width));
		sum+=mask[i];
	}
	for (int i=0;i<mask_width*mask_width;i++){
		mask[i] /= sum;
	}

	// CUDA work
	hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp,0);
    size_t gpuGlobalMem = deviceProp.totalGlobalMem;
    fprintf(stderr, "GPU global memory = %zu MBytes\n", gpuGlobalMem/(1024*1024));
    size_t freeMem, totalMem;
    hipMemGetInfo(&freeMem, &totalMem);
    fprintf(stderr, "Free = %zu MB, Total = %zu MB\n", freeMem/(1024*1024), totalMem/(1024*1024));

	unsigned char *cudaPic, *cudaOutPic;
	double *cudaMask;
	int *sizeCuda, size[3];
	size[0] = width;
	size[1] = height;
	size[2] = mask_width;
	hipMalloc((int **)&sizeCuda,3*sizeof(int));
	hipMalloc((unsigned char**)&cudaPic, width*height*3*sizeof(unsigned char));
	hipMalloc((unsigned char**)&cudaOutPic, width*height*3*sizeof(unsigned char));
	hipMalloc((double **)&cudaMask, mask_width*mask_width*sizeof(double));
	hipMemcpy(sizeCuda,size,3*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(cudaPic,pic,width*height*3*sizeof(unsigned char),hipMemcpyHostToDevice);
	hipMemcpy(cudaMask,mask,mask_width*mask_width*sizeof(double),hipMemcpyHostToDevice);
	hipMemset(cudaOutPic,0,width*height*3*sizeof(unsigned char));

	dim3 block (1024);
	dim3 grid (((width*height)/block.x)+1);
	printf("%d %d\n",block.x, grid.x);
	GaussianBlurCuda<<<grid,block>>>(cudaPic, cudaOutPic, cudaMask, sizeCuda);
	hipDeviceSynchronize();
	hipMemcpy(outpic, cudaOutPic, width*height*3*sizeof(unsigned char), hipMemcpyDeviceToHost);

	// Output file structure
	cinfo1.err = jpeg_std_error(&jerr);
	jpeg_create_compress(&cinfo1);
	jpeg_stdio_dest(&cinfo1, outfile);
	cinfo1.image_width = width;
	cinfo1.image_height = height;
	cinfo1.input_components = 3;
	cinfo1.in_color_space = JCS_RGB;
	jpeg_set_defaults(&cinfo1);
	int quality = 70;
	jpeg_set_quality(&cinfo1, quality, TRUE);
	jpeg_start_compress(&cinfo1, TRUE);

	JSAMPROW row_pointer[1];
	while(cinfo1.next_scanline < cinfo1.image_height){
		row_pointer[0] = &outpic[cinfo1.next_scanline*width*3];
		(void) jpeg_write_scanlines(&cinfo1, row_pointer, 1);
	}
	jpeg_finish_compress(&cinfo1);
	fclose(outfile);
	jpeg_destroy_compress(&cinfo1);

	double tFinish = cpuSecond();

	printf("Time elapsed: %lf seconds.\n",tFinish-tStart);
}
